#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>



#define CHECK_CUDA(x) AT_ASSERTM(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) AT_ASSERTM(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CHECK_DEVICE(x, y) AT_ASSERTM(x.device().index() == y.device().index(), #x " and " #y " must be in same CUDA device")

// C++ Python interface

// Modified from https://github.com/thomasluu/plog
template <class T>
__global__ void plog(T* input, T* output, unsigned size)
{
  unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index>=size)
    return;
  
  if (input[index] == 0.0) {
    output[index]=0.0;
    return;
  }

  x = input[index];
  T w0, w1;
  if (x > 0.0) {
    w0 = log(1.2 * x / log(2.4 * x / log1p(2.4 * x)));
  } else {
    T v = 1.4142135623730950488 * sqrt(1 + 2.7182818284590452354 * x);
    T N2 = 10.242640687119285146 + 1.9797586132081854940 * v;
    T N1 = 0.29289321881345247560 * (1.4142135623730950488 + N2);
    w0 = -1.0 + v * (N2 + v) / (N2 + v + N1 * v);
  }

  while (true) {
    T e = exp(w0);
    T f = w0 * e - x;
    w1 = w0 + ((f+f) * (1.0 + w0)) / (f * (2.0 + w0) - (e+e) * (1.0 + w0) * (1.0 + w0));
    if (fabs(w0 / w1 - 1.0) < 1.4901161193847656e-8) {
      break;
    }
    w0 = w1;
  }
  output[index]=w1
  return;
}


torch::Tensor lambertCuda(
	torch::Tensor input)
{
	CHECK_INPUT(input);

	auto output = torch::empty_like(input);

	hipSetDevice(input.device().index());

	unsigned size = 1;
	for(int i = 0; i < input.ndimension(); i++)
		size *= input.size(i);
	if(input.dtype() == torch::kFloat32)
		plog<float>(input.data<float>(), output.data<float>(), size);
	else
		plog<double>(input.data<double>(), output.data<double>(), size);
	return output;
}



PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
	m.def("lambertw", &lambertCuda, "Get lambert (CUDA)");

}